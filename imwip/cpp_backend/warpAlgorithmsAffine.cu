/*
:file:      warpAlgorithmsAffine.cu
:brief:     Affine warping algorithms
:author:    Jens Renders
*/

/*
This file is part of ImWIP.

ImWIP is free software: you can redistribute it and/or modify it under the terms of
the GNU General Public License as published by the Free Software Foundation, either
version 3 of the License, or (at your option) any later version.

ImWIP is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY;
without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
See the GNU General Public License for more details.

You should have received a copy of
the GNU General Public License along with ImWIP. If not, see <https://www.gnu.org/licenses/>.
*/

#include <stdio.h>

#include <warpKernelsAffine.cu>
#include <warpAlgorithmsAffine.hu>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void affineWarp2D(
        const float* f,
        const float* A,
        const float* b,
        float* fWarped,
        int degree,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        affineLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        affineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void adjointAffineWarp2D(
        const float* fWarped,
        const float* A,
        const float* b,
        float* f,
        int degree,
        int shape0,
        int shape1
    ){

    /*
    GPU implementation of 2D adjoint backward image warping along the DVF (u,v)
    with rectangular multivariate spline interpolation
    */


    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_A, *d_b, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    if(degree==1){
        adjointAffineLinearWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_2D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 16*16*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 16*16*sizeof(float), hipMemcpyHostToDevice));
        adjointAffineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void diffAffineWarp2D(
        const float* f,
        const float* A,
        const float* b,
        float* diffx,
        float* diffy,
        int shape0,
        int shape1
    ){

    size_t size = shape0 * shape1 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_diffx, *d_diffy;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 4 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 2 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_diffx, size));
    gpuErrchk(hipMalloc(&d_diffy, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 4 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 2 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffx, diffx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffy, diffy, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(16,16);
    dim3 numBlocks((shape0 + 15)/16, (shape1 + 15)/16);
    float coeffsx[] = {
        #include "cubic_2D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_2D_coefficients_dy.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    gpuErrchk(hipMalloc(&d_coeffsx, 16*16*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 16*16*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 16*16*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 16*16*sizeof(float), hipMemcpyHostToDevice));
    affineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffx,
        shape0,
        shape1,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp2DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffy,
        shape0,
        shape1,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(diffx, d_diffx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffy, d_diffy, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_diffx);
    hipFree(d_diffy);
}


void affineWarp3D(
        const float* f,
        const float* A,
        const float* b,
        float* fWarped,
        int degree,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_fWarped;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_fWarped, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire image
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order
    if(degree==1){
        affineLinearWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1,
            shape2
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_f,
            d_A,
            d_b,
            d_fWarped,
            shape0,
            shape1,
            shape2,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }

    // copy the result back to the host
    gpuErrchk(hipMemcpy(fWarped, d_fWarped, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void adjointAffineWarp3D(
        const float* fWarped,
        const float* A,
        const float* b,
        float* f,
        int degree,
        int shape0,
        int shape1,
        int shape2
){
    /*
    GPU implementation of 3D adjoint backward image warping along the DVF (u,v,w)
    with rectangular multivariate spline interpolation
    */

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_fWarped, *d_A, *d_b, *d_f;
    gpuErrchk(hipMalloc(&d_fWarped, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_f, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_fWarped, fWarped, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));

    // kernel invocation with 8*8*8 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order

    if(degree==1){
        adjointAffineLinearWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1,
            shape2
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }else if(degree==3){
        float coeffs[] = {
            #include "cubic_3D_coefficients.inc"
        };
        float *d_coeffs;
        gpuErrchk(hipMalloc(&d_coeffs, 64*64*sizeof(float)));
        gpuErrchk(hipMemcpy(d_coeffs, coeffs, 64*64*sizeof(float), hipMemcpyHostToDevice));
        adjointAffineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
            d_fWarped,
            d_A,
            d_b,
            d_f,
            shape0,
            shape1,
            shape2,
            d_coeffs
        );
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipFree(d_coeffs);
    }else{
        throw "Only degree 1 and 3 are implemented";
    }
    // copy the result back to the host
    gpuErrchk(hipMemcpy(f, d_f, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_fWarped);
}


void diffAffineWarp3D(
        const float* f,
        const float* A,
        const float* b,
        float* diffx,
        float* diffy,
        float* diffz,
        int shape0,
        int shape1,
        int shape2
    ){

    size_t size = shape0 * shape1 * shape2 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_diffx, *d_diffy, *d_diffz;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_diffx, size));
    gpuErrchk(hipMalloc(&d_diffy, size));
    gpuErrchk(hipMalloc(&d_diffz, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffx, diffx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffy, diffy, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffz, diffz, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order
    float coeffsx[] = {
        #include "cubic_3D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_3D_coefficients_dy.inc"
    };
    float coeffsz[] = {
        #include "cubic_3D_coefficients_dz.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    float *d_coeffsz;
    gpuErrchk(hipMalloc(&d_coeffsx, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsz, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsz, coeffsz, 64*64*sizeof(float), hipMemcpyHostToDevice));
    affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffx,
        shape0,
        shape1,
        shape2,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffy,
        shape0,
        shape1,
        shape2,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp3DKernel<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffz,
        shape0,
        shape1,
        shape2,
        d_coeffsz
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);
    hipFree(d_coeffsz);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(diffx, d_diffx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffy, d_diffy, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffz, d_diffz, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_diffx);
    hipFree(d_diffy);
    hipFree(d_diffz);
}


void diffAffineWarp3DMul(
        const float* f,
        const float* A,
        const float* b,
        float* diffx,
        float* diffy,
        float* diffz,
        int shape0,
        int shape1,
        int shape2,
        int shape3
    ){

    size_t size = shape0 * shape1 * shape2 * shape3 * sizeof(float);

    // allocate vectors in device memory
    float *d_f, *d_A, *d_b, *d_diffx, *d_diffy, *d_diffz;
    gpuErrchk(hipMalloc(&d_f, size));
    gpuErrchk(hipMalloc(&d_A, 9 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, 3 * sizeof(float)));
    gpuErrchk(hipMalloc(&d_diffx, size));
    gpuErrchk(hipMalloc(&d_diffy, size));
    gpuErrchk(hipMalloc(&d_diffz, size));

    // copy vectors from host memory to device memory
    gpuErrchk(hipMemcpy(d_f, f, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_A, A, 9 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_b, b, 3 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffx, diffx, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffy, diffy, size, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_diffz, diffz, size, hipMemcpyHostToDevice));

    // kernel invocation with 16*16 threads per block, and enough blocks
    // to cover the entire length of the vectors
    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((shape2 + 7)/8, (shape1 + 7)/8, (shape0 + 7)/8); //faster order
    float coeffsx[] = {
        #include "cubic_3D_coefficients_dx.inc"
    };
    float coeffsy[] = {
        #include "cubic_3D_coefficients_dy.inc"
    };
    float coeffsz[] = {
        #include "cubic_3D_coefficients_dz.inc"
    };
    float *d_coeffsx;
    float *d_coeffsy;
    float *d_coeffsz;
    gpuErrchk(hipMalloc(&d_coeffsx, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsy, 64*64*sizeof(float)));
    gpuErrchk(hipMalloc(&d_coeffsz, 64*64*sizeof(float)));
    gpuErrchk(hipMemcpy(d_coeffsx, coeffsx, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsy, coeffsy, 64*64*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_coeffsz, coeffsz, 64*64*sizeof(float), hipMemcpyHostToDevice));
    affineCubicWarp3DKernelMul<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffx,
        shape0,
        shape1,
        shape2,
        shape3,
        d_coeffsx
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp3DKernelMul<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffy,
        shape0,
        shape1,
        shape2,
        shape3,
        d_coeffsy
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    affineCubicWarp3DKernelMul<<<numBlocks, threadsPerBlock>>>(
        d_f,
        d_A,
        d_b,
        d_diffz,
        shape0,
        shape1,
        shape2,
        shape3,
        d_coeffsz
    );
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    hipFree(d_coeffsx);
    hipFree(d_coeffsy);
    hipFree(d_coeffsz);

    // copy the result back to the host
    gpuErrchk(hipMemcpy(diffx, d_diffx, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffy, d_diffy, size, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(diffz, d_diffz, size, hipMemcpyDeviceToHost));

    // release the device memory
    hipFree(d_f);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_diffx);
    hipFree(d_diffy);
    hipFree(d_diffz);
}